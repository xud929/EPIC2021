#include "hip/hip_runtime.h"
#include"gtrack.h"
#include<math.h>
#include<cassert>
#include<iostream>
#include<thrust/complex.h>
#include<thrust/copy.h>
#include<thrust/reduce.h>
#include<thrust/inner_product.h>
#include<thrust/transform_reduce.h>
#include<thrust/functional.h>
#include<thrust/iterator/zip_iterator.h>
#include"gwofz.h"
#include<sys/time.h>


double get_current_time(){
    struct timeval tp;
    hipDeviceSynchronize();
    gettimeofday(&tp,NULL);
    return (double)tp.tv_sec+(double)tp.tv_usec*1e-6;
}
/**************************************************************/
gThinCrabCavity::gThinCrabCavity(const ThinCrabCavity &tcc){
    phase=tcc.get_phase();
    kcc1=tcc.get_kcc();
    kcc2=kcc1*2.0;
    strength1=tcc.get_strength(1);
    strength2=tcc.get_strength(2);
}

__host__ __device__
void gThinCrabCavity::Pass(double &x, double &px, double &y, double &py, double &z, double &pz) const{
    double a1=kcc1*z+phase, a2=kcc2*z+phase, sin_t, cos_t;

    sincos(a1,&sin_t,&cos_t);
    px-=strength1*sin_t/kcc1;
    pz-=strength1*cos_t*x;

    sincos(a2,&sin_t,&cos_t);
    px-=strength2*sin_t/kcc2;
    pz-=strength2*cos_t*x;

}

/**************************************************************/
gLinearX::gLinearX(const LinearX &m){
    auto t=m.getTM();
    m11=std::get<0>(t);
    m12=std::get<1>(t);
    m21=std::get<2>(t);
    m22=std::get<3>(t);
}

__host__ __device__
void gLinearX::Pass(double &x, double &px) const{
    double u0=x, pu0=px;
    x=m11*u0+m12*pu0;
    px=m21*u0+m22*pu0;
}

/**************************************************************/
gLorentzBoost::gLorentzBoost(const LorentzBoost &m){
    auto t=m.getParams();
    cos_ang=std::get<0>(t);
    sin_ang=std::get<1>(t);
    tan_ang=std::get<2>(t);
}

gRevLorentzBoost::gRevLorentzBoost(const RevLorentzBoost &m){
    auto t=m.getParams();
    cos_ang=std::get<0>(t);
    sin_ang=std::get<1>(t);
    tan_ang=std::get<2>(t);
}

__host__ __device__
void gLorentzBoost::Pass(double &x, double &px, double &y, double &py, double &z, double &pz) const{
    double ps=1.0+pz;ps=ps*ps-px*px-py*py;ps=sqrt(ps);
    double h=1.0+pz-ps;

    py/=cos_ang;h/=cos_ang*cos_ang;
    px=px/cos_ang-h*sin_ang;
    pz-=px*sin_ang;
    ps=1.0+pz-h;

    double ds=x*sin_ang;
    x+=z*tan_ang+px/ps*ds;
    y+=py/ps*ds;
    z=z/cos_ang-h/ps*ds;
}

__host__ __device__
void gRevLorentzBoost::Pass(double &x, double &px, double &y, double &py, double &z, double &pz) const{
    double ps=1.0+pz;ps=ps*ps-px*px-py*py;ps=sqrt(ps);
    double h=1.0+pz-ps;

    x-=z*sin_ang;
    x/=1.0+(px+h*sin_ang)*sin_ang/ps;
    z=(z+h/ps*x*sin_ang)*cos_ang;
    y-=py/ps*x*sin_ang;

    pz+=px*sin_ang;
    px=(px+h*sin_ang)*cos_ang;
    py*=cos_ang;
}

/**************************************************************/
gLinear6D::gLinear6D(const Linear6D &m){
    auto t=m.getParams();
    betx=std::get<0>(t);
    bety=std::get<1>(t);
    betz=std::get<2>(t);
    alfx=std::get<3>(t);
    alfy=std::get<4>(t);
    gamx=std::get<5>(t);
    gamy=std::get<6>(t);
     xix=std::get<7>(t);
     xiy=std::get<8>(t);
     mux=std::get<9>(t);
     muy=std::get<10>(t);
     muz=std::get<11>(t);
}

__host__ __device__
void gLinear6D::Pass(double &x, double &px, double &y, double &py, double &z, double &pz) const{
    double x0=x, px0=px, y0=y, py0=py, z0=z, pz0=pz;
    double angle, cos_t, sin_t;

    angle=mux+math_const::twopi*xix*pz0;
    //cos_t=cos(angle);sin_t=sin(angle);
    sincos(angle,&sin_t,&cos_t);
    x=x0*(cos_t+alfx*sin_t)+px0*betx*sin_t;
    px=-x0*sin_t*gamx+px0*(cos_t-alfx*sin_t);

    angle=muy+math_const::twopi*xiy*pz0;
    //cos_t=cos(angle);sin_t=sin(angle);
    sincos(angle,&sin_t,&cos_t);
    y=y0*(cos_t+alfy*sin_t)+py0*bety*sin_t;
    py=-y0*sin_t*gamy+py0*(cos_t-alfy*sin_t);

#ifdef SYMPLECTIC_ONE_TURN_PASS
    double Jx=0.5*(x0*x0*gamx+2.0*alfx*x0*px0+betx*px0*px0), Jy=0.5*(y0*y0*gamy+2.0*alfy*y0*py0+bety*py0*py0);
    cos_t=cos(muz);sin_t=sin(muz);
    double tz=z0+math_const::twopi*(xix*Jx+xiy*Jy);
    z=tz*cos_t+pz0*betz*sin_t;
    pz=-tz*sin_t/betz+pz0*cos_t;
#else
    //cos_t=cos(muz);sin_t=sin(muz);
    sincos(muz,&sin_t,&cos_t);
    z=z0*cos_t+pz0*betz*sin_t;
    pz=-z0*sin_t/betz+pz0*cos_t;
#endif
}

/**************************************************************/
gGaussianStrongBeam::gGaussianStrongBeam(const GaussianStrongBeam &gsb){
    const auto &tsb=gsb.get_tsb();
    double kbb=tsb.get_slice_strength();
    xo=tsb.get_beam_centroid(0);
    yo=tsb.get_beam_centroid(1);
    auto t=tsb.get_beam_sigma();
    s11=std::get<0>(t);
    s12=std::get<1>(t);
    s22=std::get<2>(t);
    s33=std::get<3>(t);
    s34=std::get<4>(t);
    s44=std::get<5>(t);

    const auto &vc=gsb.get_slice_center();
    const auto &vw=gsb.get_slice_weight();
    size_t num_of_slices=vc.size();
    assert(num_of_slices<=MAX_SLICES);

    for(size_t i=0;i<num_of_slices;++i){
        slice_strength[num_of_slices-i-1]=vw[i]*kbb;
        slice_center[num_of_slices-i-1]=vc[i];
    }
    for(size_t i=vc.size();i<MAX_SLICES;++i){
        slice_strength[i]=nan("");
        slice_center[i]=nan("");
    }

    /*
    for(size_t i=0;i<MAX_SLICES;++i){
        std::cout<<slice_center[i]<<"\t"<<slice_strength[i]<<std::endl;
    }
    */
}

//Bassetti-Erskine formula
__host__ __device__ 
double Bassetti_Erskine(double &Kx, double &Ky, double &expterm, double sigx2, double sigy2, double x, double y){
    if(sigx2<=0.0 || sigy2<=0.0){
        Kx=0.0;Ky=0.0;expterm=1.0;
        return 0.0;
    }
    expterm=exp(-(x*x/sigx2+y*y/sigy2)/2.0);

    bool negx=(x<0), negy=(y<0), switchxy=(sigy2>sigx2);
    double sig11,sig22,sig1,sig2,x1,x2;//sig1>sig2
    if(switchxy){
        sig11=sigy2;
        sig22=sigx2;
        sig1=sqrt(sigy2);
        sig2=sqrt(sigx2);
        x1=fabs(y);
        x2=fabs(x);
    }else{
        sig11=sigx2;
        sig22=sigy2;
        sig1=sqrt(sigx2);
        sig2=sqrt(sigy2);
        x1=fabs(x);
        x2=fabs(y);
    }


    double dsize=(sig1-sig2)/2.0, msize=(sig1+sig2)/2.0;
    if(dsize/msize<round_beam_threshold){
        double temp=2.0*(1.0-expterm)/(x1*x1+x2*x2);
        Kx=temp*x;Ky=temp*y;
        return expterm/6.2831853071795862/sig1/sig2;
    }

    double denominator=1.4142135623730951*sqrt(sig11-sig22);
    auto z1=thrust::complex<double>(x1/denominator,x2/denominator), z2=thrust::complex<double>(sig2/sig1*x1/denominator,sig1/sig2*x2/denominator);
    //2*sqrt(pi)...
    auto ret=3.5449077018110318/denominator*(gEPIC::wofz(z1)-expterm*gEPIC::wofz(z2));
    Kx=(switchxy?(negx?-ret.real():ret.real()):(negx?-ret.imag():ret.imag()));
    Ky=(switchxy?(negy?-ret.imag():ret.imag()):(negy?-ret.real():ret.real()));

    return expterm/6.2831853071795862/sig1/sig2;
}

__host__ __device__ 
double gGaussianStrongBeam::Pass(double &x, double &px, double &y, double &py, double &z, double &pz) const{
    double lum=0.0;
    for(size_t i=0;i<MAX_SLICES;++i){
        double zo=slice_center[i], kbb=slice_strength[i];
        if(isnan(z) || isnan(kbb))
            break;
        double S=(z-zo)/2.0;
        double sigma11=s11+S*(s22*S-2.0*s12), sigma33=s33+S*(s44*S-2.0*s34);
        double Kx,Ky,expterm;
        lum+=Bassetti_Erskine(Kx,Ky,expterm,sigma11,sigma33,x+S*px-xo,y+S*py-yo);
        double dpx=Kx*kbb, dpy=Ky*kbb;
        px+=dpx;py+=dpy;
        x-=S*dpx;y-=S*dpy;
    }
    return lum;
}

/**************************************************************/
gtrack::gtrack(const Beam &beam, const ThinCrabCavity &ccb, const ThinCrabCavity &cca,
        const LinearX &MX1, const LinearX &MX2, const LinearX &MX3, const LinearX &MX4,
        const LorentzBoost &lb, const RevLorentzBoost &rlb, 
        const Linear6D &oneturn, const GaussianStrongBeam &gsb){

    unsigned N=beam.get_end()-beam.get_beg();
    vx.resize(N);vpx.resize(N);
    vy.resize(N);vpy.resize(N);
    vz.resize(N);vpz.resize(N);

    const auto& x=beam.get_coordinate(0);
    const auto& px=beam.get_coordinate(1);
    const auto& y=beam.get_coordinate(2);
    const auto& py=beam.get_coordinate(3);
    const auto& z=beam.get_coordinate(4);
    const auto& pz=beam.get_coordinate(5);

    thrust::copy(x.begin(),x.end(), vx.begin());
    thrust::copy(px.begin(),px.end(), vpx.begin());
    thrust::copy(y.begin(),y.end(), vy.begin());
    thrust::copy(py.begin(),py.end(), vpy.begin());
    thrust::copy(z.begin(),z.end(), vz.begin());
    thrust::copy(pz.begin(),pz.end(), vpz.begin());

    gfun.thin_crab_cavity_before_IP=gThinCrabCavity(ccb);
    gfun.thin_crab_cavity_after_IP=gThinCrabCavity(cca);
    gfun.MX1=gLinearX(MX1);gfun.MX2=gLinearX(MX2);
    gfun.MX3=gLinearX(MX3);gfun.MX4=gLinearX(MX4);
    gfun.lb=gLorentzBoost(lb);
    gfun.rlb=gRevLorentzBoost(rlb);
    gfun.oneturn=gLinear6D(oneturn);
    gfun.gsb=gGaussianStrongBeam(gsb);
}

std::vector<double> gtrack::track(unsigned n){
    auto first=thrust::make_zip_iterator(thrust::make_tuple(vx.begin(),vpx.begin(),vy.begin(),vpy.begin(),vz.begin(),vpz.begin()));
    auto last=thrust::make_zip_iterator(thrust::make_tuple(vx.end(),vpx.end(),vy.end(),vpy.end(),vz.end(),vpz.end()));

    std::vector<double> ret(18);

    gfun.turns=n;
    ret[0]=thrust::transform_reduce(first,last,gfun,0.0,thrust::plus<double>());

    ret[1]=thrust::reduce(vx.begin(),vx.end());
    ret[2]=thrust::inner_product(vx.begin(),vx.end(),vx.begin(),0.0);
    ret[3]=thrust::reduce(vpx.begin(),vpx.end());
    ret[4]=thrust::inner_product(vpx.begin(),vpx.end(),vpx.begin(),0.0);
    ret[5]=thrust::inner_product(vx.begin(),vx.end(),vpx.begin(),0.0);

    ret[6]=thrust::reduce(vy.begin(),vy.end());
    ret[7]=thrust::inner_product(vy.begin(),vy.end(),vy.begin(),0.0);
    ret[8]=thrust::reduce(vpy.begin(),vpy.end());
    ret[9]=thrust::inner_product(vpy.begin(),vpy.end(),vpy.begin(),0.0);
    ret[10]=thrust::inner_product(vy.begin(),vy.end(),vpy.begin(),0.0);

    ret[11]=thrust::reduce(vz.begin(),vz.end());
    ret[12]=thrust::inner_product(vz.begin(),vz.end(),vz.begin(),0.0);
    ret[13]=thrust::reduce(vpz.begin(),vpz.end());
    ret[14]=thrust::inner_product(vpz.begin(),vpz.end(),vpz.begin(),0.0);
    ret[15]=thrust::inner_product(vz.begin(),vz.end(),vpz.begin(),0.0);

    ret[16]=thrust::inner_product(vx.begin(),vx.end(),vz.begin(),0.0);
    ret[17]=thrust::inner_product(vy.begin(),vy.end(),vz.begin(),0.0);

    return ret;
}

/*alternate track method with less return value*/
std::vector<double> gtrack::track2(unsigned n){
    auto first=thrust::make_zip_iterator(thrust::make_tuple(vx.begin(),vpx.begin(),vy.begin(),vpy.begin(),vz.begin(),vpz.begin()));
    auto last=thrust::make_zip_iterator(thrust::make_tuple(vx.end(),vpx.end(),vy.end(),vpy.end(),vz.end(),vpz.end()));

    std::vector<double> ret(5);

    gfun.turns=n;
    ret[0]=thrust::transform_reduce(first,last,gfun,0.0,thrust::plus<double>());

    ret[1]=thrust::reduce(vx.begin(),vx.end());
    ret[2]=thrust::inner_product(vx.begin(),vx.end(),vx.begin(),0.0);

    ret[3]=thrust::reduce(vy.begin(),vy.end());
    ret[4]=thrust::inner_product(vy.begin(),vy.end(),vy.begin(),0.0);

    return ret;
}

/*alternate track method with less return value*/
void gtrack::track3(unsigned n, double *ret){
    auto first=thrust::make_zip_iterator(thrust::make_tuple(vx.begin(),vpx.begin(),vy.begin(),vpy.begin(),vz.begin(),vpz.begin()));
    auto last=thrust::make_zip_iterator(thrust::make_tuple(vx.end(),vpx.end(),vy.end(),vpy.end(),vz.end(),vpz.end()));

    gfun.turns=n;
    ret[0]=thrust::transform_reduce(first,last,gfun,0.0,thrust::plus<double>());

    ret[1]=thrust::reduce(vx.begin(),vx.end());
    ret[2]=thrust::inner_product(vx.begin(),vx.end(),vx.begin(),0.0);

    ret[3]=thrust::reduce(vy.begin(),vy.end());
    ret[4]=thrust::inner_product(vy.begin(),vy.end(),vy.begin(),0.0);
}
